#include "hip/hip_runtime.h"
%%cuda

#include<iostream>

__global__
void matrixMultiplicationKernel(float *m, float *n, float *p, int i, int j, int k) {

    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if(row < i && col < k) {
        int pValue = 0;
        for(int idx = 0; idx < j; idx++) {
            pValue += m[row * i + idx] * n[idx * i + col];
        }

        p[row * i + col] = pValue;
    }

}

int main() {

    int i = 2, j = 2, k = 2;
    float m[4] = {1,2,3,4};
    float n[4] = {5,6,7,8};
    float p[4] = {};

    float *m_d, *n_d, *p_d;

    m_d = (float*)malloc(i*j*sizeof(float));
    n_d = (float*)malloc(j*k*sizeof(float));
    p_d = (float*)malloc(i*k*sizeof(float));

    hipMalloc(&m_d, i*j*sizeof(float));
    hipMalloc(&n_d, j*k*sizeof(float));
    hipMalloc(&p_d, i*k*sizeof(float));

    hipMemcpy(m_d, m, i*j*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(n_d, n, j*k*sizeof(float), hipMemcpyHostToDevice);
    
    dim3 dimGrid = {ceil(k/16.0), ceil(i/16.0)};
    dim3 dimBlock = {16, 16};

    matrixMultiplicationKernel<<<dimGrid, dimBlock>>>(m_d, n_d, p_d, i, j, k);

    hipMemcpy(p, p_d, i*k*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(m_d);
    hipFree(n_d);
    hipFree(p_d);
    
    for(int idx = 0; idx < 4; idx++) {
        std::cout << p[idx] << " ";
    }

    return 0;
}