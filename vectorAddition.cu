
#include <hip/hip_runtime.h>
#include<iostream>

__global__
void vecAddKernel(float* A, float* B, float* C, int n) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n) {
        C[i] = A[i] + B[i];
    }
}

void vecAdd(float* A_h, float* B_h, float* C_h, int n) {
    int size = n * sizeof(float);
    float *A_d, *B_d, *C_d;
    
    hipMalloc((void**)&A_d, size);
    hipMalloc((void**)&B_d, size);
    hipMalloc((void**)&C_d, size);

    hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);

   // vecAddKernel<<<ceil(n/256.0), 256>>>(A_d, B_d, C_d, n);
    
    vecAddKernel<<<ceil(1), 2>>>(A_d, B_d, C_d, n);

    hipMemcpy(C_h, C_d, size, hipMemcpyDeviceToHost);

    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);

    printf("%f %f", *C_h, *(C_h + 1));
}

int main() {
    float A[2] = {1, 2};
    float B[2] = {3, 4};
    float C[2] = {};

    vecAdd(A, B, C, 2);
}